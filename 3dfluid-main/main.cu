#include "EventManager.h"
#include "fluid_solver.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#define SIZE 168

//define a quantidade de blocos de forma a ter metade das threads na dim i nos steps do solver
#define dx 3
//define a quantidade de blocos para usar nos steps do solver de forma a preencher as dim j,k
#define dy 84
#define dz 84
 //define as threads para a redução em 1 bloco
#define THR_RED 128
//calcula o numero total de blocos
#define numBlocks dx*dy*dz
//define os blocos que são necessários à redução em um bloco
#define AUX ((numBlocks + THR_RED - 1) / THR_RED)


#define Lblock (256)
//tamanho array max_c
#define C_size numBlocks*AUX
#define IX(i, j, k) ((i) + (M + 2) * (j) + (M + 2) * (N + 2) * (k))

// Globals for the grid size
static int M = SIZE;
static int N = SIZE;
static int O = SIZE;
static float dt = 0.1f;      // Time delta
static float diff = 0.0001f; // Diffusion constant
static float visc = 0.0001f; // Viscosity constant

// Fluid simulation arrays
static float *dens,*u,*v,*w;
float *du,*du_prev,*dv,*dv_prev,*dw,*dw_prev,*ddens,*ddens_prev;
float *d_max_c, *aux_max_c,*block_max_c;

int allocate_data_gpu() {
  int size = (M + 2) * (N + 2) * (O + 2) * sizeof(float); // Tamanho em bytes

  // Alocando memória na GPU para cada array
  hipError_t  err = hipMalloc((void**)&du, size);
  hipError_t err_v = hipMalloc((void**)&dv, size);
  hipError_t err_w = hipMalloc((void**)&dw, size);
  hipError_t err_u_prev = hipMalloc((void**)&du_prev, size);
  hipError_t err_v_prev = hipMalloc((void**)&dv_prev, size);
  hipError_t err_w_prev = hipMalloc((void**)&dw_prev, size);
  hipError_t err_dens = hipMalloc((void**)&ddens, size);
  hipError_t err_dens_prev = hipMalloc((void**)&ddens_prev, size);
  hipMalloc((void **)&d_max_c, sizeof(float));
  hipMalloc((void **)&aux_max_c, sizeof(float)*C_size);
  hipMalloc((void **)&block_max_c, sizeof(float)*Lblock);
  // Verificando se algum erro ocorreu
  if (err != hipSuccess || err_v != hipSuccess || err_w != hipSuccess ||
      err_u_prev != hipSuccess || err_v_prev != hipSuccess ||
      err_w_prev != hipSuccess || err_dens != hipSuccess ||
      err_dens_prev != hipSuccess) {

    std::cerr << "CUDA malloc failed: " << hipGetErrorString(err) << std::endl;
    return 0;
      }

  // Se todas as alocações forem bem-sucedidas
  return 1;
}
// Function to allocate simulation data
int allocate_data() {
  int size = (M + 2) * (N + 2) * (O + 2);
  dens = new float[size];
  u = new float[size];
  v = new float[size];
  w = new float[size];
  if (!dens || !u || !v || !w) {
    std::cerr << "Cannot allocate memory" << std::endl;
    return 0;
  }
  return 1;
}
// Function to clear the data (set all to zero)
void clear_data() {
  int size = (M + 2) * (N + 2) * (O + 2);
  for (int i = 0; i < size; i++) {
    dens[i] =u[i]=v[i]=w[i]= 0.0f;
  }
}
void clear_data_gpu() {
  int size = (M + 2) * (N + 2) * (O + 2) * sizeof(float); // Tamanho em bytes

  // Definindo os valores das variáveis para 0.0f na GPU
  hipError_t err;

  // Tentando limpar todas as variáveis com hipMemset
  err = hipMemset(du, 0, size);
  hipError_t err_v = hipMemset(dv, 0, size);
  hipError_t err_w = hipMemset(dw, 0, size);
  hipError_t err_u_prev = hipMemset(du_prev, 0, size);
  hipError_t err_v_prev = hipMemset(dv_prev, 0, size);
  hipError_t err_w_prev = hipMemset(dw_prev, 0, size);
  hipError_t err_dens = hipMemset(ddens, 0, size);
  hipError_t err_dens_prev = hipMemset(ddens_prev, 0, size);

  // Verificando se algum erro ocorreu após todas as chamadas de hipMemset
  if (err != hipSuccess || err_v != hipSuccess || err_w != hipSuccess ||
      err_u_prev != hipSuccess || err_v_prev != hipSuccess ||
      err_w_prev != hipSuccess || err_dens != hipSuccess ||
      err_dens_prev != hipSuccess) {

    std::cerr << "CUDA Memset failed: " << hipGetErrorString(err) << std::endl;
      }
}

// Free allocated memory
void free_data() {
  delete[] dens;
  delete[] u;
  delete[] v;
  delete[] w;
}
void free_data_gpu() {
  hipFree(du);
  hipFree(dv);
  hipFree(dw);
  hipFree(du_prev);
  hipFree(dv_prev);
  hipFree(dw_prev);
  hipFree(ddens);
  hipFree(ddens_prev);
  hipFree(aux_max_c);
  hipFree(d_max_c);
  hipFree(block_max_c);
}
// Apply events (source or force) for the current timestep

void apply_events(const std::vector<Event> &events) {
  for (const auto &event : events) {
    if (event.type == ADD_SOURCE) {
      // Apply density source at the center of the grid
      int i = M / 2, j = N / 2, k = O / 2;
      float density = event.density;
      hipMemcpy(&ddens[IX(i, j, k)], &density, sizeof(float), hipMemcpyHostToDevice);
    } else if (event.type == APPLY_FORCE) {
      // Apply forces based on the event's vector (fx, fy, fz)
      int i = M / 2, j = N / 2, k = O / 2;
      float fx = event.force.x, fy = event.force.y, fz = event.force.z;

      hipMemcpy(&du[IX(i, j, k)], &fx, sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(&dv[IX(i, j, k)], &fy, sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(&dw[IX(i, j, k)], &fz, sizeof(float), hipMemcpyHostToDevice);
    }
  }
}

// Function to sum the total density
double sum_density() {
  double total_density = 0.0;
  int size = (M + 2) * (N + 2) * (O + 2);
  for (int i = 0; i < size; i++) {
    total_density += dens[i];
  }
  return total_density;
}

// Simulation loop
void simulate(EventManager &eventManager, int timesteps) {
  int size = (M + 2) * (N + 2) * (O + 2) * sizeof(float);
  for (int t = 0; t < timesteps; t++) {
    // Get the events for the current timestep
    std::vector<Event> events = eventManager.get_events_at_timestamp(t);

    // Apply events to the simulation
    apply_events(events);
    // Perform the simulation steps
    vel_step(M, N, O, du, dv, dw, du_prev, dv_prev, dw_prev, visc, dt);
    dens_step(M, N, O, ddens, ddens_prev, du, dv, dw, diff, dt);

  }
  hipMemcpy(dens,ddens,size,hipMemcpyDeviceToHost);
}

int main() {
  // Initialize EventManager
  EventManager eventManager;
  eventManager.read_events("events.txt");

  // Get the total number of timesteps from the event file
  int timesteps = eventManager.get_total_timesteps();

  // Allocate and clear data
  //alocar memoria na GPU
  allocate_data_gpu();
  if (!allocate_data())
    return -1;
  clear_data();
  clear_data_gpu();
  // Run simulation with events
  simulate(eventManager, timesteps);

  // Print total density at the end of simulation
  double total_density = sum_density();
  std::cout << "Total density after " << timesteps
            << " timesteps: " << total_density << std::endl;

  // Free memory
  free_data();
  //libera a memoria alocada na GPU
  free_data_gpu();
  return 0;
}